#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>

  static const unsigned int c1 = 0xcc9e2d51;
  static const unsigned int c2 = 0x1b873593;
  static const unsigned int r1 = 15;
  static const unsigned int r2 = 13;
  static const unsigned int m = 5;
  static const unsigned int n = 0xe6546b64;

__device__ inline unsigned int h1(unsigned int k, unsigned int hash) {

  k *= c1;
  k = (k << r1) | (k >> (32-r1));
  k *= c2;
 
  hash ^= k;
  hash = ((hash << r2) | (hash >> (32-r2)) * m) + n;
  return hash;
}

__device__ inline unsigned int mmhash(unsigned int v1, unsigned int v2, unsigned int v3, unsigned int mod, unsigned int seed)
{
  unsigned int hash = seed;
 
  hash = h1(v1, hash);
  hash = h1(v2, hash);
  hash = h1(v3, hash);
  
  hash ^= (hash >> 16);
  hash *= 0x85ebca6b;
  hash ^= (hash >> 13);
  hash *= 0xc2b2ae35;
  hash ^= (hash >> 16);
 
  return (hash % mod);
}

#define DBSIZE (8*1024)

__global__ void __treePack(int *idata, int *treenodes, int *icats, int *jc, long long *out, int *fieldlens, 
			   int nrows, int ncols, int ntrees, int nsamps) {
  __shared__ int dbuff[DBSIZE];
  __shared__ int fl[32];
  int j, k, ic, ival;
  int seed = 45123421;

  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  if (tid < 5) {
    fl[tid] = fieldlens[tid];
  }
  __syncthreads();
  int vshift = fl[0];
  int fshift = fl[1] + vshift;
  int nshift = fl[2] + fshift;
  int tshift = fl[3] + nshift;

  int cmask = (1 << fl[0]) - 1;
  int vmask = (1 << fl[1]) - 1;
  int fmask = (1 << fl[2]) - 1;
  int nmask = (1 << fl[3]) - 1;
  int tmask = (1 << fl[4]) - 1;
  
  int nc = (DBSIZE / nrows);
  int itree = threadIdx.y;
  int jfeat = threadIdx.x;

  for (int i = nc * blockIdx.x; i < ncols; i += nc * gridDim.x) {
    int ctodo = min(nc, ncols - i);
    for (j = tid; j < nrows * ctodo; j += blockDim.x*blockDim.y) {
      dbuff[j] = idata[j + i * nrows];
    }
    __syncthreads();
    
    for (j = i; j < i + ctodo; j++) {
      for (itree = threadIdx.y; itree < ntrees; itree += blockDim.y) {
	int inode = treenodes[itree + j * ntrees];
	int ifeat = mmhash(itree, inode, jfeat, nrows, seed);
	long long hdr = (((long long)(tmask & itree)) << tshift) | (((long long)(nmask & inode)) << nshift) | (((long long)(fmask & ifeat)) << fshift);
	for (k = jc[j]; k < jc[j+1]; k++) {    
	  ic = icats[k];
	  if (jfeat < nsamps) {
	    ival = dbuff[ifeat + (j - i) * nrows];
	    out[jfeat + nsamps * (itree + ntrees * k)] = hdr | (((long long)(vmask & ival)) << vshift) | ((long long)(ic & cmask));
	  }
	}
      }
    }
    __syncthreads();
  }
}

int treePack(int *fdata, int *treenodes, int *icats, int *jc, long long *out, int *fieldlens, int nrows, int ncols, int ntrees, int nsamps) {
  int ntx = 32 * (1 + (nsamps - 1)/32);
  int nty = min(1024 / ntx, ntrees);
  dim3 bdim(ntx, nty, 1);
  int nb = min(32, 1 + (ncols-1)/32);
  __treePack<<<nb,bdim>>>(fdata, treenodes, icats, jc, out, fieldlens, nrows, ncols, ntrees, nsamps);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}


class entImpty {
 public:
  static __device__ inline float fupdate(float v) { return v * log((float)v); }
  static __device__ inline float ffinal(float vacc, float vsum) { return log(vsum) - vacc / vsum; }
};

class giniImpty {
 public:
  static __device__ inline float fupdate(float v) { return v * v; }
  static __device__ inline float ffinal(float vacc, float vsum) { return 1 - vacc / (vsum*vsum); }
};

#if __CUDA_ARCH__ >= 300

template<typename T>
__global__ void __minImpurity(long long *keys, int *counts, int *out, float *outv, int *jc, int *fieldlens, 
                              int ntrees, int nnodes, int ncats, int nsamps) {
  __shared__ int catcnt[DBSIZE];

  int tid = threadIdx.x + blockDim.x * threadIdx.y;

  if (tid < 5) {
    catcnt[tid] = fieldlens[tid];
  }
  __syncthreads();
  int vshift = catcnt[0];

  int cmask = (1 << catcnt[0]) - 1;
  int vmask = (1 << catcnt[1]) - 1;
  __syncthreads();

  int i, j, k, h, jc0, jc1, jtodo;
  long long key;
  int ccnt, ctot, cnew, cnt, ival, icat, lastival, bestival, tmp;
  float update, cacc, impty, minimpty, lastimpty, tmpx;

  for (i = threadIdx.y + blockDim.y * blockIdx.x; i < ntrees*nnodes*nsamps; i += blockDim.y * gridDim.x) {
    // Process a group with fixed itree, inode, and ifeat

    jc0 = jc[i];                                            // The range of indices for this group
    jc1 = jc[i+1];
    
    // Clear the cat counts for this group
    for (j = tid; j < DBSIZE; j += blockDim.x * blockDim.y) {
      catcnt[j] = 0;
    }
    __syncthreads();


    lastival = -1;
    lastimpty = 1e7f;
    minimpty = 1e7f;
    ctot = 0;
    cacc = 0.0f;
    for (j = jc0; j < jc1; j += blockDim.x) {
      if (j + threadIdx.x < jc1) {                         // Read a block of (32) keys and counts
        key = keys[j + threadIdx.x];                       // Each (x) thread handles a different input
        cnt = counts[j + threadIdx.x];
        icat = (int)(key & cmask);                         // Extract the cat id and integer value
        ival = ((int)(key >> vshift)) & vmask;
      }
      jtodo = min(32, jc1 - j);
      for (k = 0; k < jtodo; k++) {                        // Sequentially update counts so that each thread
        if (threadIdx.x == k) {                            // in this warp gets the old and new counts
          ccnt = catcnt[icat + ncats * threadIdx.y];       // save data for item k in thread k
          cnew = ccnt + cnt;
          catcnt[icat + ncats * threadIdx.y] = cnew;
        }
      }
      update = T::fupdate((float)cnew);                    // Compute the impurity update for this input
      if (ccnt > 0) update -= T::fupdate((float)ccnt);
#pragma unroll
      for (h = 1; h < 32; h = h + h) {                     // Form the cumsums of updates and counts
        tmpx = __shfl_up(update, h);
        tmp = __shfl_up(cnt, h);
        if (threadIdx.x >=h) {
          update += tmpx;
          cnt += tmp;
        }        
      }  
      ctot += cnt;                                        // Now update the total c and total ci log ci sums
      cacc += update;
      ctot = max(1, ctot);
      impty = T::ffinal(cacc, (float)ctot);              // And the impurity for this input

      tmp = __shfl_up(ival, 1);
      tmpx = __shfl_up(impty, 1);                         // Need the last impurity and ival in order
      if (threadIdx.x > 0) {                              // to restrict the partition feature to a value boundary
        lastival = tmp;
        lastimpty = tmpx;
      }
      if (ival == lastival) lastimpty = 1e7f;             // Eliminate values which are not at value boundaries
      if (lastimpty < minimpty) {
        minimpty = lastimpty;
        bestival = lastival;
      }

#pragma unroll
      for (h = 1; h < 32; h = h + h) {                    // Find the cumulative min impurity and corresponding ival
        tmpx = __shfl_up(minimpty, h);
        tmp = __shfl_up(bestival, h);
        if (threadIdx.x >= h && tmpx < minimpty) {
          minimpty = tmpx;
          bestival = tmp;
        }        
      }
      minimpty = __shfl(minimpty, jtodo-1);               // Carefully copy the last active thread to all threads, needed outside this loop     
      bestival = __shfl(bestival, jtodo-1);
      ctot = __shfl(ctot, jtodo-1);                
      cacc = __shfl(cacc, jtodo-1);
      lastival = __shfl(ival, jtodo-1);             
      lastimpty = __shfl(impty, jtodo-1);
    }
    if (threadIdx.x == 0) {
      out[i] = bestival;                                  // Output the best split feature value
      outv[i] = minimpty - T::ffinal(cacc, (float)ctot);  // And the impurity gain
    }
  }
}
#else
template<class T>
__global__ void __minImpurity(long long *keys, int *counts, int *out, float *outv, int *jc, int *fieldlens, 
                              int ntrees, int nnodes, int ncats, int nsamps) {}
#endif

int minImpurity(long long *keys, int *counts, int *out, float *outv, int *jc, int *fieldlens, 
                int ntrees, int nnodes, int ncats, int nsamps, int impType) {
  int ny = min(32, DBSIZE/ncats);
  dim3 tdim(32, ny, 1);
  int ng = min(64, 1L*ntrees*nnodes*nsamps);
  if (impType == 0) {
    __minImpurity<entImpty><<<ng,tdim>>>(keys, counts, out, outv, jc, fieldlens, ntrees, nnodes, ncats, nsamps);
  } else {
    __minImpurity<giniImpty><<<ng,tdim>>>(keys, counts, out, outv, jc, fieldlens, ntrees, nnodes, ncats, nsamps);
  }
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}
